
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

using namespace std;

__global__ void vecadd(const int* A, const int* B, int* C) {
	//get block index
	unsigned int block_idx = blockIdx.x;
	//get thread index
	unsigned int thread_idx = threadIdx.x;
	//get the number of threads per block
	unsigned int block_dim = blockDim.x;
	//get the thread's unique ID (block_idx * block_dim)
	unsigned int idx = (block_idx * block_dim) + thread_idx;
	//add corresponding locations of A and B and store in C
	C[idx] = A[idx] + B[idx];
}

constexpr int ELEMENTS = 1024;

int main(int argc, char** argv) {

	//create host memory
	auto data_size = sizeof(int) * ELEMENTS;
	vector<int> A(ELEMENTS);
	vector<int> B(ELEMENTS);
	vector<int> C(ELEMENTS);

	//initialize input data
	for (size_t i = 0; i < ELEMENTS; ++i) {
		A[i] = B[i] = i;
	}

	//declare buffers
	int* buff_A, * buff_B, * buff_C;

	//initialize the buffers
	hipMalloc((void**)&buff_A, data_size);
	hipMalloc((void**)&buff_B, data_size);
	hipMalloc((void**)&buff_C, data_size);

	hipMemcpy(buff_A, &A[0], data_size, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(buff_B, &B[0], data_size, hipMemcpyKind::hipMemcpyHostToDevice);

	//run kernel with one thread for each element
	//first value is number of blocks, secnod is threads per block. max 1024 threads per block
	vecadd<<<ELEMENTS / 1024, 1024>>>(buff_A, buff_B, buff_C);
	
	//wait for kernel to complete
	hipDeviceSynchronize();

	//read output buffer back to the host
	hipMemcpy(&C[0], buff_C, data_size, hipMemcpyKind::hipMemcpyDeviceToHost);
	
	for (auto element : C)
		cout << element << endl;

	//clean resources
	hipFree(buff_A);
	hipFree(buff_B);
	hipFree(buff_C);
	return 0;
}